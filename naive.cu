
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
// Kernel function to add the elements of two arrays

#define N 100000
#define BLOCKSIZE 256
__global__ void scan(int *in, int *out) {

	int gindex = threadIdx.x + blockIdx.x*blockDim.x;

	if (gindex == 0){
		out[0] = in[0];
	}
	else {	
		int sum = 0;
		for (int j = 0; j <= gindex; j++){
			sum+=in[j];
		}
		out[gindex] = sum;		
	}
//	__syncthreads();
	
}
  
double get_clock(){
	struct timeval tv; int ok;
	ok = gettimeofday(&tv, (void *) 0);
	if (ok<0) { printf("gettimeofday error"); }
	return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void)
{
	int *in, *out;
	

  // Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&in, N*sizeof(int));
  	hipMallocManaged(&out, N*sizeof(int));

  // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
    	in[i] = 1;
   		out[i] = -1;
  	}

  // Run kernel on the GPU
	int numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

	double t0 = get_clock();
	scan<<<numBlocks, BLOCKSIZE>>>(in, out);
  
	  // Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	double t1 = get_clock();
	printf("time: %f s\n", (t1-t0));
	  
	//for (int i = 0; i < N; i++){
	  //printf("%d. %d\n",i, out[i]);
  	//}

  // Free memory
  hipFree(in);
  hipFree(out);
  
  return 0;
}
