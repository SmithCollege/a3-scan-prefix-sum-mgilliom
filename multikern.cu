
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
// Kernel function to add the elements of two arrays

#define N 100000
#define BLOCKSIZE 256
#define ITEMS_PER_THREAD 100
__global__ void k1(int *in, int *k1out) {

	int gindex = threadIdx.x + blockIdx.x*blockDim.x;

	k1out[gindex*100]=in[gindex*100];
	for (int i = gindex*100+1; i <= (gindex*100 + 99); i++){ //ea thread deals w 100 items
		k1out[i]=k1out[i-1]+in[i];
	}	
}

__global__ void k3(int *k1out, int *k2out, int *k3out) {

	int gindex = threadIdx.x + blockIdx.x*blockDim.x;
	if (threadIdx.x > 0){
		for (int i = gindex*100; i <= (gindex*100 + 99); i++){ //ea thread deals w 100 items
			k3out[i]=k1out[i]+k2out[gindex-1];
		}	
	}
	else{
		for (int i = 0; i <= 99; i++){
			k1out[i]=k3out[i];
		}
	}
}



__global__ void k2(int *k1out, int *k2out) {
	k2out[0] = k1out[99];
	for (int i = 1; i < N / 100+1; i++){
		k2out[i] = k2out[i-1] + k1out[i*100+99];
	}
}




  
double get_clock(){
	struct timeval tv; int ok;
	ok = gettimeofday(&tv, (void *) 0);
	if (ok<0) { printf("gettimeofday error"); }
	return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void)
{
	int *in, *k1out, *k2out, *k3out;
//	int size_of_k2out = (N+1);
	
	
  // Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&in, N*sizeof(int));
  	hipMallocManaged(&k1out, N*sizeof(int));
  	hipMallocManaged(&k2out, N/ITEMS_PER_THREAD+1);
  	hipMallocManaged(&k3out, N*sizeof(int));

  // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
    	in[i] = 1;
   		k1out[i] = -1;
  	}

  // Run kernel on the GPU
	//int numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;
	int numThreads = N / ITEMS_PER_THREAD;
	int numBlocks = ceil(1.0 * numThreads / BLOCKSIZE);
	printf("num threads %d, numBlocks %d", numThreads, numBlocks);
	
	double t0 = get_clock();
	k1<<<numBlocks, BLOCKSIZE>>>(in, k1out);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	k2<<<1, 1>>>(k1out, k2out);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	k3<<<numBlocks, BLOCKSIZE>>>(k1out, k2out, k3out);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
  
	  // Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	double t1 = get_clock();
	printf("time: %f s\n", (t1-t0));


	for (int i = 99; i < N; i+=1000){
	  printf("%d. %d\n",i, k1out[i]);
  	}
  	#if 0
  	for (int i = 0; i < numThreads; i++){
	  printf("%d. %d\n",i, k2out[i]);
  	}

  	for (int i = 0; i < N; i+=1000){
	  printf("%d. %d\n",i, k3out[i]);
  	}
  	#endif
  	

  // Free memory
  hipFree(in);
  hipFree(k1out);
  hipFree(k2out);
  hipFree(k3out);
  
  return 0;
}
