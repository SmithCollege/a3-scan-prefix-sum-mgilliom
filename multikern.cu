
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
// Kernel function to add the elements of two arrays

#define N 400
#define BLOCKSIZE 256
__global__ void scan(int *in, int *out) {

	int gindex = threadIdx.x + blockIdx.x*blockDim.x;


	out[gindex*100]=in[gindex*100];
	for (int i = gindex*100+1; i <= (gindex*100 + 99); i++){ //ea thread deals w 100 items
		out[i]=out[i-1]+in[i];
	}
//	__syncthreads();
	
}
  
double get_clock(){
	struct timeval tv; int ok;
	ok = gettimeofday(&tv, (void *) 0);
	if (ok<0) { printf("gettimeofday error"); }
	return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void)
{
	int *in, *out;
	

  // Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&in, N*sizeof(int));
  	hipMallocManaged(&out, N*sizeof(int));

  // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
    	in[i] = 1;
   		out[i] = -1;
  	}

  // Run kernel on the GPU
	int numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

	int t0 = get_clock();
	scan<<<numBlocks, BLOCKSIZE>>>(in, out);
  
	  // Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	int t1 = get_clock();
	printf("time: %f s\n", 1000000000*(t1-t0));
	  
	for (int i = 0; i < N; i++){
	  printf("%d. %d\n",i, out[i]);
  	}

  // Free memory
  hipFree(in);
  hipFree(out);
  
  return 0;
}